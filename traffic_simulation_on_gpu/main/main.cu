#include "hip/hip_runtime.h"
/**
 *This project targets to check GPU is an option for DynaMIT.
 *This project also targets for a paper "Mesoscopic Traffic Simulation on GPU"
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "../on_cpu/network/Network.h"
#include "../on_cpu/demand/OD_Pair.h"
#include "../on_cpu/demand/OD_Path.h"
#include "../on_cpu/demand/Vehicle.h"
#include "../on_cpu/util/TimeTools.h"

#include "../on_cpu/util/shared_cpu_include.h"
#include "../on_gpu/supply/kernel_functions.h"
#include "../on_gpu/supply/OnGPUMemory.h"
#include "../on_cpu/util/SimulationResults.h"
#include "../on_gpu/supply/OnGPUVehicle.h"
#include "../on_gpu/supply/OnGPUNewLaneVehicles.h"

using namespace std;

/**
 * CUDA Execution Configuration
 */
int roadBlocks;
int roadThreadsInABlock;

int nodeBlocks;
int nodeThreadsInABlock;

/*
 * Demand
 */
Network* the_network;
vector<OD_Pair*> all_od_pairs;
vector<OD_Pair_PATH*> all_od_paths;
vector<Vehicle*> all_vehicles;

/*
 * Path Input Config
 */
std::string network_file_path = "data/network_10.dat";
std::string demand_file_path = "data/demand_10.dat";
std::string od_pair_file_path = "data/od_pair_10.dat";
std::string od_pair_paths_file_path = "data/od_pair_paths_10.dat";

/*
 * All data in GPU
 */
GPUMemory* gpu_data;

/**
 * Simulation Results
 */
std::string simulation_output_file_path = "output/simulated_outputs.txt";
std::map<int, SimulationResults*> simulation_results_pool;
ofstream simulation_results_output_file;

/*
 * GPU Streams
 * stream1: GPU Supply Simulation
 */
hipStream_t stream_gpu_supply;
hipEvent_t GPU_supply_one_time_simulation_done_event;

/*
 * Time Management
 */
long simulation_start_time;
long simulation_end_time;
long simulation_time_step;

/*
 * simulation_time is already finished time;
 * simulation_time + 1 might be the current simulating time on GPU
 */
long to_simulate_time;

/*
 * simulation_results_outputed_time is already outputted time;
 * simulation_results_outputed_time + 1 might be the outputing time on CPU
 */
long to_output_simulation_result_time;

/*
 * Define Major Functions
 */
bool load_in_network();
bool load_in_demand();
bool initilizeCPU();
bool initilizeGPU();
bool initGPUData(GPUMemory* data_local);

bool start_simulation();
bool destory_resources();

/*
 * Define Helper Functions
 */
bool copy_simulated_results_to_CPU(int time_step);
bool output_simulated_results(int time_step);

inline int timestep_to_arrayindex(int time_step) {
	return (time_step - START_TIME_STEPS) / UNIT_TIME_STEPS;
}

/*
 * Supply Function Define
 */
__global__ void supply_simulation_pre_vehicle_passing(GPUMemory* gpu_data, int time_step, int segment_length);
__global__ void supply_simulation_vehicle_passing(GPUMemory* gpu_data, int time_step, int node_length);
__global__ void supply_simulation_after_vehicle_passing(GPUMemory* gpu_data, int time_step, int segment_length);

__device__ GPUVehicle* get_next_vehicle_at_node(GPUMemory* gpu_data, int node_id, int* lane_id);

/*
 * MAIN
 */
int main() {
	if (load_in_network() == false) {
		cout << "Loading network fails" << endl;
		return 0;
	}

	if (load_in_demand() == false) {
		cout << "Loading demand fails" << endl;
		return 0;
	}

	if (initilizeCPU() == false) {
		cout << "InitilizeCPU fails" << endl;
		return 0;
	}

	if (initilizeGPU() == false) {
		cout << "InitilizeGPU fails" << endl;
		return 0;
	}

	//create streams
	hipStreamCreate(&stream_gpu_supply);
	//create a event
	hipEventCreate(&GPU_supply_one_time_simulation_done_event);

	TimeTools profile;
	profile.start_profiling();

	//Start Simulation
	if (start_simulation() == false) {
		cout << "Simulation fails" << endl;
		destory_resources();
		return 0;
	}

	profile.end_profiling();
	profile.output();

	cout << "Simulation Succeed!" << endl;

	destory_resources();
	return 0;
}

/**
 *
 */
bool load_in_network() {
	the_network = new Network();

	the_network->all_links.clear();
	the_network->all_nodes.clear();
	the_network->node_mapping.clear();

	return Network::load_network(the_network, network_file_path);
}

bool load_in_demand() {

	if (OD_Pair::load_in_all_ODs(all_od_pairs, od_pair_file_path) == false) {
		return false;
	}

	if (OD_Pair_PATH::load_in_all_OD_Paths(all_od_paths, od_pair_paths_file_path) == false) {
		return false;
	}

	if (Vehicle::load_in_all_vehicles(all_vehicles, demand_file_path) == false) {
		return false;
	}

	return true;
}

bool initilizeCPU() {
	simulation_start_time = START_TIME_STEPS;
	simulation_end_time = END_TIME_STEPS; // 2 hours
	simulation_time_step = UNIT_TIME_STEPS;

	assert(simulation_time_step == 1);

	to_simulate_time = simulation_start_time;
	to_output_simulation_result_time = simulation_start_time;

	roadThreadsInABlock = 32;
	nodeThreadsInABlock = 32;

	roadBlocks = LANE_SIZE / roadThreadsInABlock + 1;
	nodeBlocks = NODE_SIZE / nodeThreadsInABlock + 1;

	simulation_results_pool.clear();
	simulation_results_output_file.open(simulation_output_file_path.c_str());
	simulation_results_output_file << "##TIME STEP" << ":Lane ID:" << ":(" << "COUNTS" << ":" << "flow" << ":" << "density" << ":" << "speed" << ":" << "queue_length" << ")" << endl;

	return true;
}

__global__ void linkGPUData(GPUMemory *gpu_data, GPUVehicle *vpool){
	int idx = threadIdx.x * blockIdx.x * blockDim.x;
	int nVehiclePerTick = VEHICLE_MAX_LOADING_ONE_TIME * LANE_SIZE;
	GPUVehicle ***v = (GPUVehicle***)gpu_data->new_vehicles_every_time_step->new_vehicles;
}

GPUVehicle *vpool_h;
size_t vpool_size = VEHICLE_MAX_LOADING_ONE_TIME * LANE_SIZE * TOTAL_TIME_STEPS * sizeof(GPUVehicle);
bool initilizeGPU() {
	gpu_data = NULL;

	GPUMemory* data_local = new GPUMemory();
	initGPUData(data_local);
	GPUVehicle *vpool;
	printf("vpool size: %d", sizeof(GPUVehicle) * VEHICLE_MAX_LOADING_ONE_TIME * LANE_SIZE * TOTAL_TIME_STEPS);
	hipMalloc((void**)&vpool,
		sizeof(GPUVehicle) * VEHICLE_MAX_LOADING_ONE_TIME * LANE_SIZE * TOTAL_TIME_STEPS);

//	data_local->test = 1;

	if (hipMalloc(&gpu_data, data_local->total_size()) != hipSuccess) {
		cerr << "hipMalloc(&gpu_data, sizeof(GPUMemory)) failed" << endl;
	}

	/*
	 * Hi, Xiaosong, the copy fucntion needs to be changed.
	 */
	hipMemcpy(gpu_data, data_local, data_local->total_size(), hipMemcpyHostToDevice);

	// copy vpool_h to vpool, to be linked with gpu_data later. /*xiaosong*/
	hipMemcpy(vpool, vpool_h, vpool_size, hipMemcpyDeviceToHost);

	int BLOCK_SIZE = 256;
	int GRID_SIZE = TOTAL_TIME_STEPS;
	linkGPUData<<<BLOCK_SIZE, GRID_SIZE>>>(gpu_data, vpool);
	return true;
}

/*
 * Build a GPU data
 */
bool initGPUData(GPUMemory* data_local) {

	/**
	 * First Part: Lane
	 */

	for (int i = 0; i < the_network->all_links.size(); i++) {
		Link* one_link = the_network->all_links[i];

		data_local->lane_pool.lane_ID[i] = one_link->link_id;
		//make sure assert is working
//		assert(1 == 0);

		assert(one_link->link_id == i);

		data_local->lane_pool.from_node_id[i] = one_link->from_node->node_id;
		data_local->lane_pool.to_node_id[i] = one_link->to_node->node_id;

		data_local->lane_pool.Tp[i] = simulation_start_time - simulation_time_step;
		data_local->lane_pool.Tq[i] = simulation_start_time - simulation_time_step;
		data_local->lane_pool.accumulated_offset[i] = 0;

		data_local->lane_pool.flow[i] = 0;
		data_local->lane_pool.density[i] = 0;
		data_local->lane_pool.speed[i] = 0;
		data_local->lane_pool.queue_length[i] = 0;

		/*
		 * for density calculation
		 */
		data_local->lane_pool.lane_length[i] = ROAD_LENGTH; // meter
		data_local->lane_pool.max_vehicles[i] = ROAD_LENGTH / VEHICLE_LENGTH; //number of vehicles
		data_local->lane_pool.output_capacity[i] = LANE_OUTPUT_CAPACITY_TIME_STEP; //
		data_local->lane_pool.input_capacity[i] = LANE_INPUT_CAPACITY_TIME_STEP; //
		data_local->lane_pool.empty_space[i] = ROAD_LENGTH;

		/*
		 * for speed calculation
		 */
		data_local->lane_pool.alpha[i] = Alpha;
		data_local->lane_pool.beta[i] = Beta;
		data_local->lane_pool.max_density[i] = Max_Density;
		data_local->lane_pool.min_density[i] = Min_Density;
		data_local->lane_pool.MAX_SPEED[i] = MAX_SPEED;
		data_local->lane_pool.MIN_SPEED[i] = MIN_SPEED;

		data_local->lane_pool.vehicle_counts[i] = 0;
		data_local->lane_pool.vehicle_passed_to_the_lane_counts[i] = 0;

		for (int c = 0; c < MAX_VEHICLE_PER_LANE; c++) {
			data_local->lane_pool.vehicle_passed_space[c][i] = NULL;
		}

		for (int c = 0; c < LANE_INPUT_CAPACITY_TIME_STEP; c++) {
			data_local->lane_pool.vehicle_passed_space[c][i] = NULL;
		}

		for (int j = 0; j < TOTAL_TIME_STEPS; j++) {
			data_local->lane_pool.speed_history[j][i] = -1;
		}

		//it is assumed that QUEUE_LENGTH_HISTORY = 4;
		assert(QUEUE_LENGTH_HISTORY == 4);
		float weight[QUEUE_LENGTH_HISTORY];
		weight[0] = 0.5;
		weight[1] = 0.3;
		weight[2] = 0.2;
		weight[3] = 0;

		//		{ 0.2, 0.3, 0.5, 0 };

		for (int j = 0; j < QUEUE_LENGTH_HISTORY; j++) {
			data_local->lane_pool.his_queue_length[j][i] = -1;
			data_local->lane_pool.his_queue_length_weighting[j][i] = weight[j];
		}

		data_local->lane_pool.predicted_empty_space[i] = 0;
		data_local->lane_pool.predicted_queue_length[i] = 0;
	}

	/**
	 * Second Part: Node
	 */
	//	NodePool* the_node_pool = data_local->node_pool;
	for (int i = 0; i < the_network->all_nodes.size(); i++) {
		Node* one_node = the_network->all_nodes[i];

		data_local->node_pool.node_ID[i] = one_node->node_id;
		data_local->node_pool.MAXIMUM_ACCUMULATED_FLOW[i] = 0;
		data_local->node_pool.ACCUMULATYED_UPSTREAM_CAPACITY[i] = 0;
		data_local->node_pool.ACCUMULATYED_DOWNSTREAM_CAPACITY[i] = 0;

		assert(one_node->node_id == i);

		for (int j = 0; j < MAX_LANE_UPSTREAM; j++) {
			data_local->node_pool.upstream[j][i] = -1;
		}

		for (int j = 0; j < one_node->upstream_links.size(); j++) {
			data_local->node_pool.upstream[j][i] = one_node->upstream_links[j]->link_id;
			data_local->node_pool.ACCUMULATYED_UPSTREAM_CAPACITY[i] += LANE_OUTPUT_CAPACITY_TIME_STEP;
		}

		for (int j = 0; j < MAX_LANE_DOWNSTREAM; j++) {
			data_local->node_pool.downstream[j][i] = -1;
		}

		for (int j = 0; j < one_node->downstream_links.size(); j++) {
			data_local->node_pool.downstream[j][i] = one_node->downstream_links[j]->link_id;
			data_local->node_pool.ACCUMULATYED_DOWNSTREAM_CAPACITY[i] += LANE_OUTPUT_CAPACITY_TIME_STEP;
		}

		data_local->node_pool.MAXIMUM_ACCUMULATED_FLOW[i] =
				(data_local->node_pool.ACCUMULATYED_UPSTREAM_CAPACITY[i] < data_local->node_pool.ACCUMULATYED_DOWNSTREAM_CAPACITY[i]) ?
						data_local->node_pool.ACCUMULATYED_UPSTREAM_CAPACITY[i] : data_local->node_pool.ACCUMULATYED_DOWNSTREAM_CAPACITY[i];

//		std::cout << "MAXIMUM_ACCUMULATED_FLOW:" << i << ", " << data_local->node_pool.MAXIMUM_ACCUMULATED_FLOW[i] << std::endl;
	}

	/**
	 * Third Part:
	 */

	//Init VehiclePool
	for (int i = 0; i < TOTAL_TIME_STEPS; i++) {
		for (int j = 0; j < LANE_SIZE; j++) {
			data_local->new_vehicles_every_time_step[i].new_vehicle_size[j] = 0;
			data_local->new_vehicles_every_time_step[i].lane_ID[j] = -1;
		}
	}

	std::cout << "all_vehicles.size():" << all_vehicles.size() << std::endl;

	//init host vehicle pool data /*xiaosong*/
	vpool_h = (GPUVehicle*)malloc(sizeof(GPUVehicle) * VEHICLE_MAX_LOADING_ONE_TIME * LANE_SIZE * TOTAL_TIME_STEPS);
	int nVehiclePerTick = VEHICLE_MAX_LOADING_ONE_TIME * LANE_SIZE;

	//Insert Vehicles
	for (int i = 0; i < all_vehicles.size(); i++) {
		Vehicle* one_vehicle = all_vehicles[i];
//		assert(one_vehicle->vehicle_id == i);

		int time_index = one_vehicle->entry_time;
		int time_index_covert = timestep_to_arrayindex(time_index);

		assert(time_index == time_index_covert);

		int lane_ID = all_od_paths[one_vehicle->path_id]->link_ids[0];

		//try to load vehicles beyond the simulation border
		if (time_index_covert >= TOTAL_TIME_STEPS) continue;

		if (data_local->new_vehicles_every_time_step[time_index_covert]->new_vehicle_size[lane_ID] < VEHICLE_MAX_LOADING_ONE_TIME) {
			int index = data_local->new_vehicles_every_time_step[time_index_covert]->new_vehicle_size[lane_ID];
			int idx_vpool = time_index_covert * nVehiclePerTick;
			idx_vpool += index * VEHICLE_MAX_LOADING_ONE_TIME;
			idx_vpool += lane_ID;

			vpool_h[idx_vpool].vehicle_ID = one_vehicle->vehicle_id;
			vpool_h[idx_vpool].entry_time = time_index;
			vpool_h[idx_vpool].current_lane_ID = lane_ID;
			int max_copy_length =
				MAX_ROUTE_LENGTH > all_od_paths[one_vehicle->path_id]->link_ids.size() ?
				all_od_paths[one_vehicle->path_id]->link_ids.size() :
				MAX_ROUTE_LENGTH;

			for (int p = 1; p < max_copy_length; p++) {
				vpool_h[idx_vpool].path_code[p - 1] = all_od_paths[one_vehicle->path_id]->route_code[p] ? 1 : 0;
			}

			//ready for the next lane, so next_path_index is set to 1, if the next_path_index == whole_path_length, it means cannot find path any more, can exit;
			vpool_h[idx_vpool].next_path_index = 1;
			vpool_h[idx_vpool].whole_path_length = all_od_paths[one_vehicle->path_id]->link_ids.size();

			data_local->new_vehicles_every_time_step[time_index_covert]->new_vehicle_size[lane_ID]++;
		}
		else {
			std::cout << "Loading Vehicles Exceeds The Loading Capacity: Time:" << time_index_covert << ", Lane_ID:" << lane_ID << std::endl;
		}
	}

	//test
//	for (int i = 0; i < TOTAL_TIME_STEPS; i++) {
//		int new_size = 0;
//
//		for (int j = 0; j < LANE_SIZE; j++) {
//			new_size += data_local->new_vehicles_every_time_step[i]->new_vehicle_size[j];
//		}
//
//		std::cout << "new_size: AT " << i << ", " << new_size << std::endl;
//	}

//	data_local->test = 126;

	return true;
}

bool destory_resources() {
	simulation_results_output_file.flush();
	simulation_results_output_file.close();

	hipEventDestroy(GPU_supply_one_time_simulation_done_event);
	hipStreamDestroy(stream_gpu_supply);
	return true;
}

bool start_simulation() {
	bool first_time_step = true;

	/*
	 * Simulation Loop
	 */

	while (((to_simulate_time >= simulation_end_time) && (to_output_simulation_result_time >= simulation_end_time)) == false) {

		//GPU has done simulation at current time
		if (to_simulate_time < simulation_end_time && (hipEventQuery(GPU_supply_one_time_simulation_done_event) == hipSuccess)) {
			//step 1
			if (first_time_step == true) {
				first_time_step = false;
			}
			else {
				copy_simulated_results_to_CPU(to_simulate_time);
				to_simulate_time += simulation_time_step;
			}

			//step 2
			cout << "to_simulate_time:" << to_simulate_time << ", simulation_end_time:" << simulation_end_time << endl;

			//setp 3
			supply_simulation_pre_vehicle_passing<<<roadBlocks, roadThreadsInABlock, 0, stream_gpu_supply>>>(gpu_data, to_simulate_time, LANE_SIZE);
			supply_simulation_vehicle_passing<<<nodeBlocks, nodeThreadsInABlock, 0, stream_gpu_supply>>>(gpu_data, to_simulate_time, NODE_SIZE);
			supply_simulation_after_vehicle_passing<<<roadBlocks, roadThreadsInABlock, 0, stream_gpu_supply>>>(gpu_data, to_simulate_time, LANE_SIZE);

			hipEventRecord(GPU_supply_one_time_simulation_done_event, stream_gpu_supply);
		}
		//GPU is busy, so CPU does something else (I/O)
		else if (to_output_simulation_result_time < to_simulate_time) {
			output_simulated_results(to_output_simulation_result_time);
			to_output_simulation_result_time += simulation_time_step;
		}
		else {
			cout << "---------------------" << endl;
			cout << "CPU nothing to do" << endl;
			cout << "to_simulate_time:" << to_simulate_time << endl;
			cout << "to_output_simulation_result_time:" << to_output_simulation_result_time << endl;
			cout << "---------------------" << endl;
		}
	}

	return true;
}

/**
 * Minor Functions
 */
bool copy_simulated_results_to_CPU(int time_step) {
	int index = timestep_to_arrayindex(time_step);
	SimulationResults* one = new SimulationResults();

	hipMemcpy(one->flow, gpu_data->lane_pool.flow, sizeof(float) * LANE_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(one->density, gpu_data->lane_pool.density, sizeof(float) * LANE_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(one->speed, gpu_data->lane_pool.speed, sizeof(float) * LANE_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(one->queue_length, gpu_data->lane_pool.queue_length, sizeof(float) * LANE_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(one->counts, gpu_data->lane_pool.vehicle_counts, sizeof(int) * LANE_SIZE, hipMemcpyDeviceToHost);

	simulation_results_pool[index] = one;
	return true;
}

bool output_simulated_results(int time_step) {
	if (simulation_results_pool.find(time_step) == simulation_results_pool.end()) {
		std::cerr << "System Error, Try to output time " << time_step << ", while it is not ready!" << std::endl;
		return false;
	}

	int index = timestep_to_arrayindex(time_step);
	SimulationResults* one = simulation_results_pool[index];
	assert(one != NULL);

	for (int i = 0; i < LANE_SIZE; i++) {
		simulation_results_output_file << time_step << ":lane:" << i << ":(" << one->counts[i] << ":" << one->flow[i] << ":" << one->density[i] << ":" << one->speed[i] << ":" << one->queue_length[i]
				<< ")" << endl;
	}

//	temply not deleted
//	if(one != NULL)
//		delete one;

	return true;
}

/**
 * Kernel Functions, not sure how to move to other folder
 */

/*
 * Supply Function Implementation
 */
__global__ void supply_simulation_pre_vehicle_passing(GPUMemory* gpu_data, int time_step, int segment_length) {
	int lane_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (lane_id >= segment_length) return;

	int time_index = time_step;

	gpu_data->lane_pool.new_vehicle_join_counts[lane_id] = 0;

	//init capacity
	gpu_data->lane_pool.input_capacity[lane_id] = LANE_INPUT_CAPACITY_TIME_STEP;
	gpu_data->lane_pool.output_capacity[lane_id] = LANE_OUTPUT_CAPACITY_TIME_STEP;

	//init for next GPU kernel function
	gpu_data->lane_pool.blocked[lane_id] = false;

	//load passed vehicles to the back of the lane
	for (int i = 0; i < gpu_data->lane_pool.vehicle_passed_to_the_lane_counts[lane_id]; i++) {
		if (gpu_data->lane_pool.vehicle_counts[lane_id] < gpu_data->lane_pool.max_vehicles[lane_id]) {
			gpu_data->lane_pool.vehicle_space[gpu_data->lane_pool.vehicle_counts[lane_id]][lane_id] = gpu_data->lane_pool.vehicle_passed_space[i][lane_id];
			gpu_data->lane_pool.vehicle_counts[lane_id]++;

			gpu_data->lane_pool.new_vehicle_join_counts[lane_id]++;
		}
	}
	gpu_data->lane_pool.vehicle_passed_to_the_lane_counts[lane_id] = 0;

	//
	//load newly generated vehicles to the back of the lane
	for (int i = 0; i < gpu_data->new_vehicles_every_time_step[time_index]->new_vehicle_size[lane_id]; i++) {
		if (gpu_data->lane_pool.vehicle_counts[lane_id] < gpu_data->lane_pool.max_vehicles[lane_id]) {
			gpu_data->lane_pool.vehicle_space[gpu_data->lane_pool.vehicle_counts[lane_id]][lane_id] = &(gpu_data->new_vehicles_every_time_step[time_index]->new_vehicles[i][lane_id]);
			gpu_data->lane_pool.vehicle_counts[lane_id]++;

			gpu_data->lane_pool.new_vehicle_join_counts[lane_id]++;
		}
	}

	//update speed and density
	gpu_data->lane_pool.density[lane_id] = 1.0 * VEHICLE_LENGTH * gpu_data->lane_pool.vehicle_counts[lane_id] / gpu_data->lane_pool.lane_length[lane_id];

	//Speed-Density Relationship
	gpu_data->lane_pool.speed[lane_id] = gpu_data->lane_pool.MAX_SPEED[lane_id]
			* (pow((1 - pow((gpu_data->lane_pool.density[lane_id] / gpu_data->lane_pool.max_density[lane_id]), gpu_data->lane_pool.beta[lane_id])), gpu_data->lane_pool.alpha[lane_id]));

	if (gpu_data->lane_pool.speed[lane_id] < gpu_data->lane_pool.MIN_SPEED[lane_id]) gpu_data->lane_pool.speed[lane_id] = gpu_data->lane_pool.MIN_SPEED[lane_id];

	//update speed history
	gpu_data->lane_pool.speed_history[time_index][lane_id] = gpu_data->lane_pool.speed[lane_id];

	//estimated empty_space
	if (time_step < START_TIME_STEPS + 4 * UNIT_TIME_STEPS) {
//		gpu_data->lane_pool.predicted_empty_space[lane_id] = gpu_data->lane_pool.his_queue_length[0][lane_id];
		gpu_data->lane_pool.predicted_queue_length[lane_id] = 0;
		gpu_data->lane_pool.predicted_empty_space[lane_id] = ROAD_LENGTH;
	}
	else {
		gpu_data->lane_pool.predicted_queue_length[lane_id] = gpu_data->lane_pool.his_queue_length[0][lane_id];
		gpu_data->lane_pool.predicted_queue_length[lane_id] += (gpu_data->lane_pool.his_queue_length[0][lane_id] - gpu_data->lane_pool.his_queue_length[1][lane_id])
				* gpu_data->lane_pool.his_queue_length_weighting[0][lane_id];

		gpu_data->lane_pool.predicted_queue_length[lane_id] += (gpu_data->lane_pool.his_queue_length[1][lane_id] - gpu_data->lane_pool.his_queue_length[2][lane_id])
				* gpu_data->lane_pool.his_queue_length_weighting[1][lane_id];

		gpu_data->lane_pool.predicted_queue_length[lane_id] += (gpu_data->lane_pool.his_queue_length[2][lane_id] - gpu_data->lane_pool.his_queue_length[3][lane_id])
				* gpu_data->lane_pool.his_queue_length_weighting[2][lane_id];

		//need improve
		//XUYAN, need modify
		gpu_data->lane_pool.predicted_empty_space[lane_id] = (ROAD_LENGTH - gpu_data->lane_pool.predicted_queue_length[lane_id]);
	}

	//update Tp
	gpu_data->lane_pool.accumulated_offset[lane_id] += gpu_data->lane_pool.speed[lane_id] * UNIT_TIME_STEPS; //meter

	while (gpu_data->lane_pool.accumulated_offset[lane_id] >= gpu_data->lane_pool.lane_length[lane_id]) {
		gpu_data->lane_pool.accumulated_offset[lane_id] -= gpu_data->lane_pool.speed_history[gpu_data->lane_pool.Tp[lane_id]][lane_id] * UNIT_TIME_STEPS;
		gpu_data->lane_pool.Tp[lane_id] += UNIT_TIME_STEPS;
	}
}

__global__ void supply_simulation_vehicle_passing(GPUMemory* gpu_data, int time_step, int node_length) {
	int node_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (node_id >= node_length) return;

	for (int i = 0; i < gpu_data->node_pool.MAXIMUM_ACCUMULATED_FLOW[node_id]; i++) {
		int lane_id = -1;

		//Find A vehicle
		GPUVehicle* one_v = get_next_vehicle_at_node(gpu_data, node_id, &lane_id);

		if (one_v == NULL || lane_id < 0) {
//			printf("one_v == NULL\n");
			break;
		}

		//Insert to next Lane
		if (gpu_data->lane_pool.vehicle_space[0][lane_id]->next_path_index >= gpu_data->lane_pool.vehicle_space[0][lane_id]->whole_path_length) {
			//the vehicle has finished the trip

//			printf("vehicle %d finish trip at node %d,\n", one_v->vehicle_ID, node_id);
		}
		else {
			int next_lane_index = gpu_data->lane_pool.vehicle_space[0][lane_id]->path_code[gpu_data->lane_pool.vehicle_space[0][lane_id]->next_path_index];
			int next_lane_id = gpu_data->node_pool.downstream[next_lane_index][node_id];
			gpu_data->lane_pool.vehicle_space[0][lane_id]->next_path_index++;

			//it is very critical to update the entry time when passing
			gpu_data->lane_pool.vehicle_space[0][lane_id]->entry_time = time_step;

			//add the vehicle
			gpu_data->lane_pool.vehicle_passed_space[gpu_data->lane_pool.vehicle_passed_to_the_lane_counts[next_lane_id]][next_lane_id] = one_v;
			gpu_data->lane_pool.vehicle_passed_to_the_lane_counts[next_lane_id]++;

			gpu_data->lane_pool.input_capacity[next_lane_id]--;
			gpu_data->lane_pool.predicted_empty_space[next_lane_id] -= VEHICLE_LENGTH;

//			printf("time_step=%d,one_v->vehicle_ID=%d,lane_id=%d, next_lane_id=%d, next_lane_index=%d\n", time_step, one_v->vehicle_ID, lane_id, next_lane_id, next_lane_index);
		}

		//Remove from current Lane
		for (int j = 1; j < gpu_data->lane_pool.vehicle_counts[lane_id]; j++) {
			gpu_data->lane_pool.vehicle_space[j - 1][lane_id] = gpu_data->lane_pool.vehicle_space[j][lane_id];
		}

		gpu_data->lane_pool.vehicle_counts[lane_id]--;
		gpu_data->lane_pool.output_capacity[lane_id]--;
		gpu_data->lane_pool.flow[lane_id]++;
	}
}

__device__ GPUVehicle* get_next_vehicle_at_node(GPUMemory* gpu_data, int node_id, int* lane_id) {

	int maximum_waiting_time = -1;
//	int the_lane_id = -1;

	for (int j = 0; j < MAX_LANE_UPSTREAM; j++) {

		int one_lane_id = gpu_data->node_pool.upstream[j][node_id];
		if (one_lane_id < 0) continue;

		/*
		 * Condition 1: The Lane is not NULL
		 * ----      2: Has Output Capacity
		 * ---       3: Is not blocked
		 * ---       4: Has vehicles
		 * ---       5: The vehicle can pass
		 */

		if (gpu_data->lane_pool.output_capacity[one_lane_id] > 0 && gpu_data->lane_pool.blocked[one_lane_id] == false && gpu_data->lane_pool.vehicle_counts[one_lane_id] > 0) {
			int time_diff = gpu_data->lane_pool.Tp[one_lane_id] - gpu_data->lane_pool.vehicle_space[0][one_lane_id]->entry_time;
			if (time_diff >= 0) {

				//if already the final move, then no need for checking next road
				if ((gpu_data->lane_pool.vehicle_space[0][one_lane_id]->next_path_index) >= (gpu_data->lane_pool.vehicle_space[0][one_lane_id]->whole_path_length)) {
					if (time_diff > maximum_waiting_time) {
						maximum_waiting_time = time_diff;
						*lane_id = one_lane_id;
						return gpu_data->lane_pool.vehicle_space[0][one_lane_id];
					}
				}
				else {
					int next_lane_index = gpu_data->lane_pool.vehicle_space[0][one_lane_id]->path_code[gpu_data->lane_pool.vehicle_space[0][one_lane_id]->next_path_index];
					int next_lane_id = gpu_data->node_pool.downstream[next_lane_index][node_id];

					/**
					 * Condition 6: The Next Lane has input capacity
					 * ---       7: The next lane has empty space
					 */
					if (gpu_data->lane_pool.input_capacity[next_lane_id] > 0 && gpu_data->lane_pool.predicted_empty_space[next_lane_id] > VEHICLE_LENGTH) {
						if (time_diff > maximum_waiting_time) {
							maximum_waiting_time = time_diff;
							*lane_id = one_lane_id;
							return gpu_data->lane_pool.vehicle_space[0][one_lane_id];
						}
					}
					else {
						gpu_data->lane_pool.blocked[one_lane_id] = true;
					}
				}
			}
		}
	}

	return NULL;
}

__global__ void supply_simulation_after_vehicle_passing(GPUMemory* gpu_data, int time_step, int segment_length) {
	int lane_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (lane_id >= segment_length) return;

	//update queue length
	bool continue_loop = true;
	float queue_length = 0;
	float acc_length_moving = gpu_data->lane_pool.accumulated_offset[lane_id];
	int to_time_step = gpu_data->lane_pool.Tp[lane_id];

	for (int i = 0; continue_loop && i < gpu_data->lane_pool.vehicle_counts[lane_id]; i++) {
		if (gpu_data->lane_pool.vehicle_space[i][lane_id]->entry_time <= gpu_data->lane_pool.Tp[lane_id]) {
			queue_length += VEHICLE_LENGTH;
		}
		else {
			int entry_time = gpu_data->lane_pool.vehicle_space[i][lane_id]->entry_time;
			for (int j = entry_time; i < to_time_step; i++) {
				acc_length_moving -= gpu_data->lane_pool.speed_history[j][lane_id] * UNIT_TIME_STEPS;
			}

			if (acc_length_moving + queue_length >= gpu_data->lane_pool.lane_length[lane_id]) {
				to_time_step = entry_time;
				queue_length += VEHICLE_LENGTH;
			}
			else {
				continue_loop = false;
			}
		}
	}

	//update queue length
	gpu_data->lane_pool.queue_length[lane_id] = queue_length;

	//update the queue history
	for (int i = 3; i > 0; i--) {
		gpu_data->lane_pool.his_queue_length[i][lane_id] = gpu_data->lane_pool.his_queue_length[i - 1][lane_id];
	}
	gpu_data->lane_pool.his_queue_length[0][lane_id] = queue_length;

	//update the empty space
	if (gpu_data->lane_pool.new_vehicle_join_counts[lane_id] > 0) {
		gpu_data->lane_pool.empty_space[lane_id] = gpu_data->lane_pool.speed[lane_id] * UNIT_TIME_STEPS - gpu_data->lane_pool.new_vehicle_join_counts[lane_id] * VEHICLE_LENGTH;
		if (gpu_data->lane_pool.empty_space[lane_id] < 0) gpu_data->lane_pool.empty_space[lane_id] = 0;
	}
	else {
		gpu_data->lane_pool.empty_space[lane_id] = gpu_data->lane_pool.empty_space[lane_id] + gpu_data->lane_pool.speed[lane_id] * UNIT_TIME_STEPS;
	}
}
